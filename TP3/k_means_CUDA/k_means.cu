#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>

typedef struct Point
{
    float x, y;
} Point;

__global__ void compute_distances(Point *points, Point *clusters, int *count, float *dists, int *indices, const int N, const int K)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N)
        return;
    float min_value = 10000;
    int min_index = 0;

    for (int i = 0; i < K; i++)
    {
        float distx = points[tid].x - clusters[i].x;
        float disty = points[tid].y - clusters[i].y;
        float dist = distx * distx + disty * disty;
        if (dist < min_value)
        {
            min_value = dist;
            min_index = i;
        }
    }

    dists[tid] = min_value;
    indices[tid] = min_index;
}

__global__ void update_clusters(Point *clusters, int *count, float *sum_dist_x, float *sum_dist_y, const int K)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= K)
        return;
    float x = sum_dist_x[tid] / count[tid];
    float y = sum_dist_y[tid] / count[tid];

    clusters[tid].x = x;
    clusters[tid].y = y;
}

void init(Point *points, Point *clusters, const int N, const int K)
{
    srand(10);

    for (int i = 0; i < N; i++)
    {
        points[i].x = (float)rand() / RAND_MAX;
        points[i].y = (float)rand() / RAND_MAX;
    }

    for (int i = 0; i < K; i++)
    {
        clusters[i].x = points[i].x;
        clusters[i].y = points[i].y;
    }
}

int kmeans(Point *points, Point *clusters, int *count, const int N, const int K)
{
    int changed = 0;
    int *indices;
    float *dists;
    hipMalloc((void **)&indices, N * sizeof(int));
    hipMalloc((void **)&dists, N * sizeof(float));

    int block_size = 32;
    int num_blocks = (N + block_size - 1) / block_size;
    compute_distances<<<num_blocks, block_size>>>(points, clusters, count, dists, indices, N, K);

    hipMemcpy(indices, indices, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(dists, dists, N * sizeof(float), hipMemcpyDeviceToHost);

    float sum_dist_x[K];
    float sum_dist_y[K];
    for (int i = 0; i < K; i++)
    {
        count[i] = 0;
        sum_dist_x[i] = 0;
        sum_dist_y[i] = 0;
    }
    for (int i = 0; i < N; i++)
    {
        int min_index = indices[i];
        count[min_index]++;
        sum_dist_x[min_index] += points[i].x;
        sum_dist_y[min_index] += points[i].y;
    }

    hipMemcpy(sum_dist_x, sum_dist_x, K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sum_dist_y, sum_dist_y, K * sizeof(float), hipMemcpyHostToDevice);

    update_clusters<<<num_blocks, block_size>>>(clusters, count, sum_dist_x, sum_dist_y, K);

    hipMemcpy(clusters, clusters, K * sizeof(Point), hipMemcpyDeviceToHost);

    for (int i = 0; i < K; i++)
    {
        if (clusters[i].x != clusters[i].x || clusters[i].y != clusters[i].y)
        {
            changed = 1;
            break;
        }
    }

    hipFree(indices);
    hipFree(dists);

    return changed;
}

int main(int argc, char **argv)
{
    if (argc < 4)
        return -1;
    const int N = atoi(argv[1]);
    const int K = atoi(argv[2]);

    Point *points = malloc(N * sizeof(Point));
    Point *clusters = malloc(K * sizeof(Point));
    int *count = malloc(K * sizeof(Point));
    int iterator = 0;

    hipError_t error = hipSuccess;
    error = hipSetDevice(0);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to set device: %s\n", hipGetErrorString(error));
        return -1;
    }

    init(points, clusters, N, K);

    do
    {
        iterator++;
    } while (kmeans(points, clusters, count, N, K) && iterator < 20);

    printf("N = %d, K = %d\n", N, K);
    for (int i = 0; i < K; i++)
    {
        printf("Center: (%.3f, %.3f) %d\n", clusters[i].x, clusters[i].y, count[i]);
    }
    printf("Iterations: %d times \n", iterator);

    free(points);
    free(clusters);
    free(count);

    return 0;
}